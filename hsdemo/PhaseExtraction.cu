#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <omp.h>
#include "PhaseExtraction.cuh"
#include "DCT_Unwrapping.cuh"
using namespace std;

/*-------------------------------------------------------------------------*/
/* This computes an in-place complex-to-complex FFT                        */
/* x and y are the real and imaginary arrays of 2^m points.                */
/* dir =  1 gives forward transform                                        */
/* dir = -1 gives reverse transform                                        */
/*                                                                         */
/*  Formula: forward                                                       */
/*                N-1                                                      */
/*                ---                                                      */
/*            1   \          - j k 2 pi n / N                              */
/*    X(n) = ---   >   x(k) e                    = forward transform       */
/*            N   /                                n=0..N-1                */
/*                ---                                                      */
/*                k=0                                                      */
/*                                                                         */
/*    Formula: reverse                                                     */
/*                N-1                                                      */
/*                ---                                                      */
/*                \          j k 2 pi n / N                                */
/*    X(n) =       >   x(k) e                    = forward transform       */
/*                /                                n=0..N-1                */
/*                ---                                                      */
/*                k=0                                                      */
/*-------------------------------------------------------------------------*/
int FFT(int dir, int m, double *x, double *y)
{
	long nn, i, i1, j, k, i2, l, l1, l2;
	double c1, c2, tx, ty, t1, t2, u1, u2, z;

	/* Calculate the number of points */
	nn = 1;
	for (i = 0; i<m; i++)
		nn *= 2;
	/* Do the bit reversal */
	i2 = nn >> 1;
	j = 0;
	for (i = 0; i<nn - 1; i++)
	{
		if (i < j)
		{
			tx = x[i];
			ty = y[i];
			x[i] = x[j];
			y[i] = y[j];
			x[j] = tx;
			y[j] = ty;
		}
		k = i2;
		while (k <= j)
		{
			j -= k;
			k >>= 1;
		}
		j += k;
	}

	/* Compute the FFT */
	c1 = -1.0;
	c2 = 0.0;
	l2 = 1;
	for (l = 0; l<m; l++)
	{
		l1 = l2;
		l2 <<= 1;
		u1 = 1.0;
		u2 = 0.0;
		for (j = 0; j<l1; j++)
		{
			for (i = j; i<nn; i += l2)
			{
				i1 = i + l1;
				t1 = u1 * x[i1] - u2 * y[i1];
				t2 = u1 * y[i1] + u2 * x[i1];
				x[i1] = x[i] - t1;
				y[i1] = y[i] - t2;
				x[i] += t1;
				y[i] += t2;
			}
			z = u1 * c1 - u2 * c2;
			u2 = u1 * c2 + u2 * c1;
			u1 = z;
		}
		c2 = sqrt((1.0 - c1) / 2.0);
		if (dir == 1)
			c2 = -c2;
		c1 = sqrt((1.0 + c1) / 2.0);
	}

	/* Scaling for forward transform */
	if (dir == 1)
	{
		for (i = 0; i<nn; i++)
		{
			x[i] /= (double)nn;
			y[i] /= (double)nn;
		}
	}
	return(true);
}
/*-------------------------------------------------------------------------*/
int DFT(int dir, int m, double *x1, double *y1)
{
	long i, k;
	double arg;
	double cosarg, sinarg;

	double *x2 = (double *)malloc(m*sizeof(double));
	double *y2 = (double *)malloc(m*sizeof(double));

	if (x2 == NULL || y2 == NULL)
		return(false);

	for (i = 0; i<m; i++) {
		x2[i] = 0;
		y2[i] = 0;
		arg = -dir * 2.0 * M_PI * (double)i / (double)m;
		for (k = 0; k<m; k++) {
			cosarg = cos(k * arg);
			sinarg = sin(k * arg);
			x2[i] += (x1[k] * cosarg - y1[k] * sinarg);
			y2[i] += (x1[k] * sinarg + y1[k] * cosarg);
		}
	}

	/* Copy the data back */
	if (dir == 1) {
		for (i = 0; i<m; i++) {
			x1[i] = x2[i] / (double)m;
			y1[i] = y2[i] / (double)m;
		}
	}
	else {
		for (i = 0; i<m; i++) {
			x1[i] = x2[i];
			y1[i] = y2[i];
		}
	}

	free(x2);
	free(y2);
	return(true);
}
/*-------------------------------------------------------------------------*/
/* Butterworth filter                                                      */
/*-------------------------------------------------------------------------*/
void bfilter(complex<float> *filter, int width, int height)
{
	for (int v = 0; v<height; v++)
	for (int u = 0; u<width; u++)
	{
		int pos = u + v*width;
		float temp_v = (u - width / 2)*(u - width / 2) + (v - height / 2)*(v - height / 2);
		double distance = sqrt(temp_v);
		double H = 1 / (1 + pow(distance / 2.0, 0.1));
		filter[u + v*width] = complex<float>(real(filter[u + v*width])*H, imag(filter[u + v*width])*H);
	}
}
/*-------------------------------------------------------------------------*/
void FFT1Dshift(complex<float> *input, int length)
{
	complex<float> tmp;

	for (int i = 0; i < length / 2; i++)
	{
		tmp = input[i];
		input[i] = input[i + length / 2];
		input[i + length / 2] = tmp;
	}
}
/*-------------------------------------------------------------------------*/
__global__ void cuFFT1Dshift(hipfftComplex *input, int width)
{
	hipfftComplex tmp;

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i<width / 2)
	{
		// interchange entries in 4 quadrants, 1 <--> 3 and 2 <--> 4
		tmp = input[i];
		input[i] = input[i + width / 2];
		input[i + width / 2] = tmp;
	}
}
/*-------------------------------------------------------------------------*/
/*Ref.: goo.gl/DR9Pqs*/
void FFT2Dshift(complex<float> *input, int width, int height)
{
	complex<float> tmp13, tmp24;

	// interchange entries in 4 quadrants, 1 <--> 3 and 2 <--> 4
	for (int k = 0; k < height / 2; k++)
	for (int i = 0; i < width / 2; i++)
	{
		tmp13 = input[i + k*width];
		input[i + k*width] = input[(i + width / 2) + (k + height / 2)*width];
		input[(i + width / 2) + (k + height / 2)*width] = tmp13;
		tmp24 = input[(i + width / 2) + k*width];
		input[(i + width / 2) + k*width] = input[i + (k + height / 2)*width];
		input[i + (k + height / 2)*width] = tmp24;
	}
}
/*-------------------------------------------------------------------------*/
__global__ void cuFFT2Dshift(hipfftComplex *input, int width, int height)
{
	hipfftComplex tmp13, tmp24;

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (i<width / 2 && j<height / 2)
	{
		// interchange entries in 4 quadrants, 1 <--> 3 and 2 <--> 4
		tmp13 = input[i + j*width];
		input[i + j*width] = input[(i + width / 2) + (j + height / 2)*width];
		input[(i + width / 2) + (j + height / 2)*width] = tmp13;
		tmp24 = input[(i + width / 2) + j*width];
		input[(i + width / 2) + j*width] = input[i + (j + height / 2)*width];
		input[i + (j + height / 2)*width] = tmp24;
	}
}
/*-------------------------------------------------------------------------*/
/*Ref.: goo.gl/3ZEKgN*/
void FFT3Dshift(complex<float> *input, int width, int height, int slice)
{
	complex<float> tmp1, tmp2, tmp3, tmp4;

	for (int k = 0; k < slice / 2; k++)
	for (int j = 0; j < height / 2; j++)
	for (int i = 0; i < width / 2; i++)
	{
		tmp1 = input[i + j*width + k*width*height];
		input[i + j*width + k*width*height] = input[(width / 2 + i) + (height / 2 + j)*width + (slice / 2 + k)*width*height];
		input[(width / 2 + i) + (height / 2 + j)*width + (slice / 2 + k)*width*height] = tmp1;

		tmp2 = input[i + (height / 2 + j)*width + k*width*height];
		input[i + (height / 2 + j)*width + k*width*height] = input[(width / 2 + i) + j*width + (slice / 2 + k)*width*height];
		input[(width / 2 + i) + j*width + (slice / 2 + k)*width*height] = tmp2;

		tmp3 = input[(width / 2 + i) + j*width + k*width*height];
		input[(width / 2 + i) + j*width + k*width*height] = input[i + (height / 2 + j)*width + (slice / 2 + k)*width*height];
		input[i + (height / 2 + j)*width + (slice / 2 + k)*width*height] = tmp3;

		tmp4 = input[(width / 2 + i) + (height / 2 + j)*width + k*width*height];
		input[(width / 2 + i) + (height / 2 + j)*width + k*width*height] = input[i + j*width + (slice / 2 + k)*width*height];
		input[i + j*width + (slice / 2 + k)*width*height] = tmp4;
	}
}
/*-------------------------------------------------------------------------*/
void FFT3Dshift_cufftComplex(hipfftComplex *input, int width, int height, int slice)
{
	hipfftComplex tmp1, tmp2, tmp3, tmp4;

	for (int k = 0; k < slice / 2; k++)
	for (int j = 0; j < height / 2; j++)
	for (int i = 0; i < width / 2; i++)
	{
		tmp1 = input[i + j*width + k*width*height];
		input[i + j*width + k*width*height] = input[(width / 2 + i) + (height / 2 + j)*width + (slice / 2 + k)*width*height];
		input[(width / 2 + i) + (height / 2 + j)*width + (slice / 2 + k)*width*height] = tmp1;

		tmp2 = input[i + (height / 2 + j)*width + k*width*height];
		input[i + (height / 2 + j)*width + k*width*height] = input[(width / 2 + i) + j*width + (slice / 2 + k)*width*height];
		input[(width / 2 + i) + j*width + (slice / 2 + k)*width*height] = tmp2;

		tmp3 = input[(width / 2 + i) + j*width + k*width*height];
		input[(width / 2 + i) + j*width + k*width*height] = input[i + (height / 2 + j)*width + (slice / 2 + k)*width*height];
		input[i + (height / 2 + j)*width + (slice / 2 + k)*width*height] = tmp3;

		tmp4 = input[(width / 2 + i) + (height / 2 + j)*width + k*width*height];
		input[(width / 2 + i) + (height / 2 + j)*width + k*width*height] = input[i + j*width + (slice / 2 + k)*width*height];
		input[i + j*width + (slice / 2 + k)*width*height] = tmp4;
	}
}
/*-------------------------------------------------------------------------*/
__global__ void cuFFT3Dshift(hipfftComplex *input, int width, int height, int slice)
{
	hipfftComplex tmp1, tmp2, tmp3, tmp4;

	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int k = blockDim.z * blockIdx.z + threadIdx.z;

	if ((i<width / 2) && (j<height / 2) && (k<slice / 2))
	{
		tmp1 = input[i + j*width + k*width*height];
		input[i + j*width + k*width*height] = input[(width / 2 + i) + (height / 2 + j)*width + (slice / 2 + k)*width*height];
		input[(width / 2 + i) + (height / 2 + j)*width + (slice / 2 + k)*width*height] = tmp1;

		tmp2 = input[i + (height / 2 + j)*width + k*width*height];
		input[i + (height / 2 + j)*width + k*width*height] = input[(width / 2 + i) + j*width + (slice / 2 + k)*width*height];
		input[(width / 2 + i) + j*width + (slice / 2 + k)*width*height] = tmp2;

		tmp3 = input[(width / 2 + i) + j*width + k*width*height];
		input[(width / 2 + i) + j*width + k*width*height] = input[i + (height / 2 + j)*width + (slice / 2 + k)*width*height];
		input[i + (height / 2 + j)*width + (slice / 2 + k)*width*height] = tmp3;

		tmp4 = input[(width / 2 + i) + (height / 2 + j)*width + k*width*height];
		input[(width / 2 + i) + (height / 2 + j)*width + k*width*height] = input[i + j*width + (slice / 2 + k)*width*height];
		input[i + j*width + (slice / 2 + k)*width*height] = tmp4;
	}
}
/*-------------------------------------------------------------------------*/
int Powerof2(int n, int *m, int *twopm)
{
	if (n <= 1)
	{
		*m = 0;
		*twopm = 1;
		return(false);
	}

	*m = 1;
	*twopm = 2;
	do{
		(*m)++;
		(*twopm) *= 2;
	} while (2 * (*twopm) <= n);

	if (*twopm != n)
		return(false);
	else
		return(true);
}
/*-------------------------------------------------------------------------*/
/* Perform a 2D FFT inplace given a complex 2D array                       */
/* The direction dir, 1 for forward, -1 for reverse                        */
/* The size of the array (nx,ny)                                           */
/* Return false if there are memory problems or                            */
/*    the dimensions are not powers of 2                                   */
/*-------------------------------------------------------------------------*/
int FFT2D(complex<float> *c, int nx, int ny, int dir)
{
	int m, twopm;
	double *realC, *imagC;

	/* Transform the rows */
	realC = (double *)malloc(nx * sizeof(double));
	imagC = (double *)malloc(nx * sizeof(double));
	if (realC == NULL || imagC == NULL)
		return(false);
	if (!Powerof2(nx, &m, &twopm) || twopm != nx)
		return(false);
	for (int j = 0; j<ny; j++)
	{
		for (int i = 0; i<nx; i++)
		{
			realC[i] = (double)real(c[i*ny + j]);
			imagC[i] = (double)imag(c[i*ny + j]);
		}

		FFT(dir, m, realC, imagC);

		for (int i = 0; i<nx; i++)
		{
			c[i*ny + j] = complex<float>((float)realC[i], (float)imagC[i]);
		}
	}

	/* Transform the columns */
	realC = (double *)realloc(realC, nx * sizeof(double));
	imagC = (double *)realloc(imagC, nx * sizeof(double));
	if (realC == NULL || imagC == NULL)
		return(false);
	if (!Powerof2(ny, &m, &twopm) || twopm != ny)
		return(false);
	for (int i = 0; i<nx; i++)
	{
		for (int j = 0; j<ny; j++)
		{
			realC[j] = (double)real(c[i*ny + j]);
			imagC[j] = (double)imag(c[i*ny + j]);
		}

		FFT(dir, m, realC, imagC);

		for (int j = 0; j<ny; j++)
		{
			c[i*ny + j] = complex<float>((float)realC[j], (float)imagC[j]);
		}
	}
	free(realC);
	free(imagC);

	return(true);
}
/*-------------------------------------------------------------------------*/
void FFT3D(complex<float> *c, int nx, int ny, int nz, int dir)
{
#pragma omp parallel for
	for (int z = 0; z<nz; z++)
	{
		complex <float> *temp_f = (complex<float> *)malloc(nx*ny*sizeof(complex<float>));

		for (int i = 0; i < nx*ny; i++)
		{
			temp_f[i] = c[i + z*nx*ny];
		}

		FFT2D(temp_f, nx, ny, dir);
		for (int i = 0; i < nx*ny; i++)
		{
			c[i + z*nx*ny] = temp_f[i];
		}
		free(temp_f);
	}
#pragma omp barrier

	//int m,twopm;	
	//double *realC, *imagC;

	// Transform the rows

	//Transform the slices
#pragma omp parallel for
	for (int i = 0; i<nx*ny; i++)
	{
		double *realC = (double *)malloc(nz * sizeof(double));
		double *imagC = (double *)malloc(nz * sizeof(double));

		int m, twopm;
		Powerof2(nz, &m, &twopm);

		for (int k = 0; k<nz; k++)
		{
			realC[k] = (double)real(c[k*nx*ny + i]);
			imagC[k] = (double)imag(c[k*nx*ny + i]);
		}

		FFT(dir, m, realC, imagC);

		for (int k = 0; k<nz; k++)
		{
			c[k*nx*ny + i] = complex<float>((float)realC[k], (float)imagC[k]);
		}
		free(realC);
		free(imagC);
	}
#pragma omp barrier

}


void cuFFT1D(hipfftComplex *ImgArray, int size, int batch, int dir)
{
	//Create a 1D FFT plan. 
	hipfftHandle plan;
	hipfftPlan1d(&plan, size, HIPFFT_C2C, batch);

	if (dir == -1)
	{
		// Use the CUFFT plan to transform the signal out of place. 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_FORWARD);

		//		hipDeviceSynchronize();
	}
	else if (dir == 1)
	{
		// Note: idata != odata indicates an out-of-place transformation to CUFFT at execution time. 
		//Inverse transform the signal in place 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_BACKWARD);

		hipDeviceSynchronize();

		int grid = (size + 1024 - 1) / 1024;
		int block = 32 * 32;
		scaleFFT1D << <grid, block >> >(ImgArray, size, 1.f / size);
	}
	else if (dir == 2)
	{
		// Note: idata != odata indicates an out-of-place transformation to CUFFT at execution time. 
		//Inverse transform the signal in place 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_BACKWARD);

		hipDeviceSynchronize();
	}

	// Destroy the CUFFT plan.
	hipfftDestroy(plan);
}
void cuFFT2D(hipfftComplex *ImgArray, int sizeX, int sizeY, int dir)
{
	//Create a 2D FFT plan. 
	hipfftHandle plan;
	hipfftPlan2d(&plan, sizeX, sizeY, HIPFFT_C2C);
	/*const int NRANK = 2;
	const int BATCH = 10;

	int n [NRANK] = {sizeX, sizeY} ;
	hipfftPlanMany(&plan , 2 , n ,
	NULL , 1 , sizeX*sizeY ,
	NULL , 1 , sizeX*sizeY ,
	HIPFFT_C2C , BATCH );*/


	if (dir == -1)
	{
		// Use the CUFFT plan to transform the signal out of place. 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_FORWARD);

		hipDeviceSynchronize();
	}
	else if (dir == 1)
	{
		// Note: idata != odata indicates an out-of-place transformation to CUFFT at execution time. 
		//Inverse transform the signal in place 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_BACKWARD);

		hipDeviceSynchronize();

		int blocksInX = (sizeX + 32 - 1) / 32;
		int blocksInY = (sizeY + 32 - 1) / 32;
		dim3 grid(blocksInX, blocksInY);
		dim3 block(32, 32);
		scaleFFT2D << <grid, block >> >(ImgArray, sizeX, sizeY, 1.f / (sizeX*sizeY));
	}

	// Destroy the CUFFT plan.
	hipfftDestroy(plan);
}

void cuFFT2D_Batch(hipfftComplex *ImgArray, int sizeX, int sizeY, int sizeZ, int dir)
{
	//Create a 2D FFT plan. 
	hipfftHandle plan;
	const int NRANK = 2;
	const int BATCH = sizeZ;

	int n[NRANK] = { sizeX, sizeY };
	hipfftPlanMany(&plan, 2, n,
		NULL, 1, sizeX*sizeY,
		NULL, 1, sizeX*sizeY,
		HIPFFT_C2C, BATCH);


	if (dir == -1)
	{
		// Use the CUFFT plan to transform the signal out of place. 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_FORWARD);

		hipDeviceSynchronize();
	}
	else if (dir == 1)
	{
		// Note: idata != odata indicates an out-of-place transformation to CUFFT at execution time. 
		//Inverse transform the signal in place 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_BACKWARD);

		hipDeviceSynchronize();

		int blocksInX = (sizeX + 32 - 1) / 32;
		int blocksInY = (sizeY + 32 - 1) / 32;
		dim3 grid(blocksInX, blocksInY);
		dim3 block(32, 32);
		//scaleFFT2D << <grid, block >> >(ImgArray, sizeX, sizeY, 1.f / (sizeX*sizeY));
	}

	// Destroy the CUFFT plan.
	hipfftDestroy(plan);
}

void cuFFT2Dz(hipfftDoubleComplex *ImgArray, int sizeX, int sizeY, int dir)
{
	//Create a 2D FFT plan. 
	hipfftHandle plan;
	hipfftPlan2d(&plan, sizeX, sizeY, HIPFFT_Z2Z);	//cufftSafeCall(hipfftPlan2d(&plan, sizeX, sizeY, HIPFFT_C2C));


	if (dir == -1)
	{
		// Use the CUFFT plan to transform the signal out of place. 
		hipfftExecZ2Z(plan, (hipfftDoubleComplex *)ImgArray, (hipfftDoubleComplex *)ImgArray, HIPFFT_FORWARD);
	}
	else if (dir == 1)
	{
		// Note: idata != odata indicates an out-of-place transformation to CUFFT at execution time. 
		//Inverse transform the signal in place 
		hipfftExecZ2Z(plan, (hipfftDoubleComplex *)ImgArray, (hipfftDoubleComplex *)ImgArray, HIPFFT_BACKWARD);

		int blocksInX = (sizeX + 32 - 1) / 32;
		int blocksInY = (sizeY + 32 - 1) / 32;
		dim3 grid(blocksInX, blocksInY);
		dim3 block(32, 32);
		scaleFFT2Dz << <grid, block >> >(ImgArray, sizeX, sizeY, 1.f / (sizeX*sizeY));
	}

	// Destroy the CUFFT plan.
	hipfftDestroy(plan);
}

void cuFFT3D(hipfftComplex *ImgArray, int sizeX, int sizeY, int sizeZ, int dir)
{
	//Create a 3D FFT plan. 
	hipfftHandle plan;
	hipfftPlan3d(&plan, sizeX, sizeY, sizeZ, HIPFFT_C2C);
	//int batch = 10;
	//int dims[] = {sizeZ, sizeY, sizeX}; // reversed order
	//hipfftPlanMany(&plan, 3, dims, NULL, 1, 0, NULL, 1, 0, HIPFFT_C2C, batch);

	if (dir == -1)
	{
		// Use the CUFFT plan to transform the signal out of place. 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_FORWARD);
	}
	else if (dir == 1)
	{
		// Note: idata != odata indicates an out-of-place transformation to CUFFT at execution time. 
		//Inverse transform the signal in place 
		hipfftExecC2C(plan, (hipfftComplex *)ImgArray, (hipfftComplex *)ImgArray, HIPFFT_BACKWARD);

		int blocksInX = (sizeX + 8 - 1) / 8;
		int blocksInY = (sizeY + 8 - 1) / 8;
		int blocksInZ = (sizeZ + 8 - 1) / 8;
		dim3 grid(blocksInX, blocksInY, blocksInZ);
		dim3 block(8, 8, 8);

		scaleFFT3D << <grid, block >> >(ImgArray, sizeX, sizeY, sizeZ, 1.f / (sizeX*sizeY*sizeZ));
	}

	// Destroy the CUFFT plan.
	hipfftDestroy(plan);
}

__global__ void scaleFFT1D(hipfftComplex *cu_F, int nx, double scale)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

	if (xIndex<nx)
	{
		double tempX = cu_F[xIndex].x * scale;
		double tempY = cu_F[xIndex].y * scale;
		cu_F[xIndex].x = (float)tempX;
		cu_F[xIndex].y = (float)tempY*(-1);
	}
}

__global__ void scaleFFT2D(hipfftComplex *cu_F, int nx, int ny, double scale)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if ((xIndex<nx) && (yIndex<ny))
	{
		unsigned int index_out = xIndex + nx*yIndex;
		double tempX = cu_F[index_out].x * scale;
		double tempY = cu_F[index_out].y * scale;
		cu_F[index_out].x = (float)tempX;
		cu_F[index_out].y = (float)tempY*(-1);
	}
}

__global__ void scaleFFT2Dz(hipfftDoubleComplex *cu_F, int nx, int ny, double scale)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if ((xIndex<nx) && (yIndex<ny))
	{
		unsigned int index_out = xIndex + nx*yIndex;
		double tempX = cu_F[index_out].x * scale;
		double tempY = cu_F[index_out].y * scale;
		cu_F[index_out].x = (double)tempX;
		cu_F[index_out].y = (double)tempY*(-1);
	}
}


__global__ void scaleFFT2DReal(float *cu_F, int nx, int ny, double scale)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if ((xIndex<nx) && (yIndex<ny))
	{
		unsigned int index_out = xIndex + nx*yIndex;
		double tempX = cu_F[index_out] * scale;
		cu_F[index_out] = (float)tempX;
	}
}


__global__ void scaleFFT3D(hipfftComplex *cu_F, int nx, int ny, int nz, double scale)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	if ((xIndex<nx) && (yIndex<ny) && (zIndex<nz))
	{
		unsigned int index_out = xIndex + nx*yIndex + nx*ny*zIndex;
		double tempX = cu_F[index_out].x * scale;
		double tempY = cu_F[index_out].y * scale;
		cu_F[index_out].x = (float)tempX;
		cu_F[index_out].y = (float)tempY;
	}
}



void cuFFT_Real(hipfftComplex *freq, float *img, const unsigned int Nx, const unsigned int Ny, int dir)
{
	size_t   Ny_pad = ((Ny >> 1) + 1);
	//size_t   Ny_pad = Ny;
	size_t   N_pad = Nx * Ny_pad;
	size_t   stride = 2 * Ny_pad; // stride on real data	

	// step 1: transfer data to device, sequence by sequence
	hipfftReal *img_plane;
	hipMalloc((void**)&img_plane, sizeof(hipfftReal)*Nx*Ny);
	hipMemcpy(img_plane, img, sizeof(hipfftReal)*Nx*Ny, hipMemcpyDeviceToDevice);

	hipfftComplex *FFT_plane;
	hipMalloc((void**)&FFT_plane, sizeof(hipfftComplex)*Nx*Ny_pad);
	hipMemcpy(FFT_plane, freq, sizeof(hipfftComplex)*Nx*Ny_pad, hipMemcpyDeviceToDevice);

	// step 2: Create a 2D FFT plan. 
	// step 3: Use the CUFFT plan to transform the signal in-place.
	hipfftHandle plan;
	hipfftResult flag;
	if (dir == -1)
	{
		hipfftPlan2d(&plan, Nx, Ny, HIPFFT_R2C);
		flag = hipfftExecR2C(plan, (hipfftReal*)img_plane, (hipfftComplex*)FFT_plane);

		hipMemcpy(freq, FFT_plane, sizeof(hipfftComplex)*Nx*Ny_pad, hipMemcpyDeviceToDevice);
	}
	else if (dir == 1)
	{
		hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2R);
		flag = hipfftExecC2R(plan, (hipfftComplex*)FFT_plane, (hipfftReal*)img_plane);

		int blocksInX = (Nx + 32 - 1) / 32;
		int blocksInY = (Ny + 32 - 1) / 32;
		dim3 grid(blocksInX, blocksInY);
		dim3 block(32, 32);
		scaleFFT2DReal << <grid, block >> >(img_plane, Nx, Ny, 1.f / (Nx*Ny));

		hipMemcpy(img, img_plane, sizeof(hipfftReal)*Nx*Ny, hipMemcpyDeviceToDevice);
	}

	if (flag != HIPFFT_SUCCESS)	printf("2D: cufftExec fails\n");

	// make sure that all threads are done
	hipDeviceSynchronize();

	// step 4: copy data to host
	//hipMemcpy(h_idata, d_idata, sizeof(hipfftComplex)*N_pad, hipMemcpyDeviceToHost);

	// Destroy the CUFFT plan.
	hipfftDestroy(plan);
	hipFree(FFT_plane);
	hipFree(img_plane);
}

//--------------------------------------------------------------------------------------
__global__ void bilinear_interpolation_kernel(float * __restrict__ d_result, const float * __restrict__ d_data,
	const int M1, const int N1, const int M2, const int N2)
{
	const int i = threadIdx.x + blockDim.x * blockIdx.x;
	const int j = threadIdx.y + blockDim.y * blockIdx.y;

	const float x_ratio = ((float)(M1 - 1)) / M2;
	const float y_ratio = ((float)(N1 - 1)) / N2;

	if ((i<M2) && (j<N2))
	{
		float result_temp1, result_temp2;

		const int    ind_x = (int)(x_ratio * i);
		const float  a = (x_ratio * i) - ind_x;

		const int    ind_y = (int)(y_ratio * j);
		const float  b = (y_ratio * j) - ind_y;

		float d00, d01, d10, d11;
		if (((ind_x)   < M1) && ((ind_y)   < N1))  d00 = d_data[ind_y    *M1 + ind_x];	else	d00 = 0.f;
		if (((ind_x + 1) < M1) && ((ind_y)   < N1))  d10 = d_data[ind_y    *M1 + ind_x + 1];	else	d10 = 0.f;
		if (((ind_x)   < M1) && ((ind_y + 1) < N1))  d01 = d_data[(ind_y + 1)*M1 + ind_x];	else	d01 = 0.f;
		if (((ind_x + 1) < M1) && ((ind_y + 1) < N1))  d11 = d_data[(ind_y + 1)*M1 + ind_x + 1];	else	d11 = 0.f;

		result_temp1 = a * d10 + (-d00 * a + d00);
		result_temp2 = a * d11 + (-d01 * a + d01);
		d_result[i + M2*j] = b * result_temp2 + (-result_temp1 * b + result_temp1);
	}
}
//--------------------------------------------------------------------------------------
void sequence1DFFT(float *ResampleArray, hipfftComplex *out_array, int Nx, int Ny)
{
	//host memory
	//hipfftComplex *host_FFT = (hipfftComplex *)malloc(Nx * (Ny / 4) * sizeof(hipfftComplex));
	//hipfftComplex *host_out = (hipfftComplex *)malloc((Nx / 4)*(Ny / 4) *sizeof(hipfftComplex));
	//device memory
	hipfftComplex *device_FFT, *out_FFT;
	float *sumFFT_1D;
	hipMalloc((void **)&device_FFT, sizeof(hipfftComplex)*Nx*(Ny / 4));
	hipMalloc((void **)&out_FFT, sizeof(hipfftComplex)*(Nx / 4)*(Ny / 4));
	hipMalloc((void **)&sumFFT_1D, sizeof(float)*Nx);
	hipMemset(sumFFT_1D, 0, Nx*sizeof(float));

	//copy the floating array to hipfftComplex
	dim3 dimGrid(Nx / TILE_DIM, Ny / 4 / TILE_DIM, 1);
	dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
	real2cufft << <dimGrid, dimBlock >> >(device_FFT, ResampleArray);

	//1D FFT
	cuFFT1D(device_FFT, Nx, Ny / 4, -1);
	//DeviceMemOutFFT("D:\\device_FFT.1024.256.raw", device_FFT, Nx, Ny / 4);
	//crop the component from FT domain
	int blocksInX = (Nx + 32 - 1) / 32;
	int blocksInY = (Ny / 4 + 32 - 1) / 32;
	dim3 grid(blocksInX, blocksInY);
	dim3 block(32, 32);
	shiftArray << <grid, block >> >(device_FFT, Nx, Ny / 4);
	HistogramFT << <grid, block >> >(sumFFT_1D, device_FFT, Nx, Ny / 4);
	//DeviceMemOut("D:\\sumFFT_1D.1024.1.raw", sumFFT_1D, Nx, 1);
	//find out the maximum and its index
	thrust::device_ptr<float> max_ptr = thrust::device_pointer_cast(sumFFT_1D);
	thrust::device_ptr<float> result_offset = thrust::max_element(max_ptr + int(Nx*0.6), max_ptr + Nx);

	//float max_value = result_offset[0];
	int max_idx = &result_offset[0] - &max_ptr[0];
	//printf("\nMininum value = %f\n", max_value);
	//printf("Position = %i\n", &result_offset[0] - &max_ptr[0]);

	int blocksX2 = (Nx / 4 + 32 - 1) / 32;
	int blocksY2 = (Ny / 4 + 32 - 1) / 32;
	dim3 grid2(blocksX2, blocksY2);
	dim3 block2(32, 32);
	CropFTdomain << <grid2, block2 >> >(device_FFT, out_FFT, Nx, Ny, max_idx);
	//DeviceMemOutFFT("D:\\out_FFT.256.256.raw", out_FFT, (Nx / 4), (Ny / 4));
	shiftArray << <grid, block >> >(out_FFT, Nx / 4, Ny / 4);
	//inverse FFT
	cuFFT1D(out_FFT, Nx / 4, Ny / 4, 1);
	//DeviceMemOutFFT("D:\\out_iFFT.256.256.raw", out_FFT, (Nx / 4), (Ny / 4));
	dim3 dimGrid2(Nx / 4 / TILE_DIM, Ny / 4 / TILE_DIM, 1);
	dim3 dimBlock2(TILE_DIM, BLOCK_ROWS, 1);
	copySharedMem << <dimGrid2, dimBlock2 >> >(out_array, out_FFT, Nx / 4);

	hipFree(device_FFT);
	hipFree(sumFFT_1D);
	hipFree(out_FFT);
	//free(host_out);
	//free(host_FFT);
}
//--------------------------------------------------------------------------------------
__global__ void real2cufft(hipfftComplex *odata, const float *idata)
{
	__shared__ float tile[TILE_DIM * TILE_DIM];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		tile[(threadIdx.y + j)*TILE_DIM + threadIdx.x] = idata[(y + j)*width + x];

	__syncthreads();

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
	{
		odata[(y + j)*width + x].x = tile[(threadIdx.y + j)*TILE_DIM + threadIdx.x];
		odata[(y + j)*width + x].y = 0;
	}


}
//--------------------------------------------------------------------------------------
__global__ void copySharedMem(hipfftComplex *odata, const hipfftComplex *idata, const float scale)
{
	__shared__ hipfftComplex tile[TILE_DIM][TILE_DIM];

	int x = blockIdx.x * TILE_DIM + threadIdx.x;
	int y = blockIdx.y * TILE_DIM + threadIdx.y;
	int width = gridDim.x * TILE_DIM;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
		tile[threadIdx.y + j][threadIdx.x] = idata[(y + j)*width + x];

	__syncthreads();

	x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
	y = blockIdx.x * TILE_DIM + threadIdx.y;

	for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
	{
		odata[(y + j)*width + x].x = tile[threadIdx.x][threadIdx.y + j].x * (1 / scale);
		odata[(y + j)*width + x].y = tile[threadIdx.x][threadIdx.y + j].y * (-1 / scale);
	}

}
//--------------------------------------------------------------------------------------
__global__ void HistogramFT(float *sumFFT_1D, hipfftComplex *device_FFT, int Nx, int Ny)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if ((xIndex < Nx) && (yIndex < Ny))
	{
		unsigned int idx = xIndex + Nx*yIndex;
		sumFFT_1D[xIndex] += log10(sqrt(device_FFT[idx].x*device_FFT[idx].x + device_FFT[idx].y*device_FFT[idx].y));
	}
}
//--------------------------------------------------------------------------------------
__global__ void shiftArray(hipfftComplex *device_FFT, int Nx, int Ny)
{
	hipfftComplex tmp;

	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;

	if (xIndex<Nx / 2 && yIndex<Ny)
	{
		tmp = device_FFT[xIndex + Nx*yIndex];
		device_FFT[xIndex + Nx*yIndex] = device_FFT[(xIndex + Nx / 2) + Nx*yIndex];
		device_FFT[(xIndex + Nx / 2) + Nx*yIndex] = tmp;
	}
}
//--------------------------------------------------------------------------------------
__global__ void CropFTdomain(hipfftComplex *device_FFT, hipfftComplex *device_crop, int Nx, int Ny, int center)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int idx1, idx2;

	if (xIndex < Nx / 4 && yIndex < Ny / 4)
	{
		idx1 = xIndex + (Nx / 4)*yIndex;
		idx2 = (xIndex - (Nx / 8) + center) + Nx*yIndex;
		device_crop[idx1] = device_FFT[idx2];
	}
}

//--------------------------------------------------------------------------------------
__global__ void convertFFT2float(float *dst, hipfftComplex *src, int Nx, int Ny)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int idx;

	if (xIndex < Nx && yIndex < Ny)
	{
		idx = xIndex + Nx * yIndex;
		dst[idx] = src[idx].x;
	}
}
//--------------------------------------------------------------------------------------
__global__ void convert2oneByte(uint8_t *dst, float *src, float max, float min, float range, int Nx, int Ny)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int idx = yIndex*Nx + xIndex;
	
	if (xIndex < Nx && yIndex < Ny)
	{
		dst[idx] = (uint8_t)(255.0 * (src[idx] - min) / range);
	}
}
//--------------------------------------------------------------------------------------
void extractQPI(float *SP, hipfftComplex *cuSP_FFT, int Nx, int Ny)
{
	int blocksInX = (Nx + 32 - 1) / 32;
	int blocksInY = (Ny + 32 - 1) / 32;
	dim3 grid(blocksInX, blocksInY);
	dim3 block(32, 32);

	float *cuSP_temp, *cuSP_resample;
	hipMalloc((void **)&cuSP_temp, sizeof(float)*Nx *Ny);
	hipMalloc((void **)&cuSP_resample, sizeof(float)*Nx*(Ny / 4));

	hipMemcpy(cuSP_temp, SP, sizeof(float)*Nx*Ny, hipMemcpyHostToDevice);

	bilinear_interpolation_kernel << <grid, block >> >(cuSP_resample, cuSP_temp, Nx, Ny, Nx, Ny / 4);

	sequence1DFFT(cuSP_resample, cuSP_FFT, Nx, Ny);

	hipFree(cuSP_temp);
	hipFree(cuSP_resample);
}

__global__ void estimateWrapPhase(float *SPWrap, hipfftComplex *SP, int sizeX, int sizeY)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int i;

	if ((xIndex<sizeX) && (yIndex<sizeY))
	{
		i = xIndex + sizeX * yIndex;
		SPWrap[i] = atan2(SP[i].y, SP[i].x);
	}
}

//--------------------------------------------------------------------------------------
__global__ void estimatePhase(float *Phase, float *UnSPWrap, int sizeX, int sizeY)
{
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int idx;
	if ((xIndex<sizeX) && (yIndex<sizeY))
	{
		idx = xIndex + sizeX * yIndex;
		Phase[idx] = (float)(UnSPWrap[idx]);

		if (isnan(Phase[idx]) || isinf(Phase[idx]))
			Phase[idx] = 0;
	}
}

void FastExtraction(uint8_t *dst, uint8_t *src, int Nx, int Ny)
{
	//for original size
	int blocksInX = (Nx + 32 - 1) / 32;
	int blocksInY = (Ny + 32 - 1) / 32;
	dim3 grid(blocksInX, blocksInY);
	dim3 block(32, 32);

	//for original size/4
	int blocksInX3 = (Nx/4 + 32 - 1) / 32;
	int blocksInY3 = (Ny/4 + 32 - 1) / 32;
	dim3 grid3(blocksInX3, blocksInY3);
	dim3 block3(32, 32);

	
	
#pragma omp parallel for
	for (int j = 0; j < Ny; j++)
		for (int i = 0; i < Nx; i++) 
		{
			SP_float[j + i*Nx] = (float)src[i + j*Nx];
		}	
#pragma omp barrier

	extractQPI(SP_float, cuSP2, Nx, Ny);
	//DeviceMemOutFFT("D:\\cuSP2.256.256.raw", cuSP2, Nx2, Ny2);
	
	estimateWrapPhase << <grid3, block3 >> >(SPWrapPhase2, cuSP2, Nx2, Ny2);
	//DeviceMemOut("D:\\SPWrapPhase2.256.256.raw", SPWrapPhase2, Nx2, Ny2);
	//UWLS
	estimatePhase << <grid3, block3 >> >(UnWrapPhaseSP2, SPWrapPhase2, Nx2, Ny2);
	DCT_UWLS_Unwrapped(cuPhaseMap2, UnWrapPhaseSP2, Nx2, Ny2);
	//DeviceMemOut("D:\\UnWrapPhaseSP2.256.256.raw", UnWrapPhaseSP2, Nx2, Ny2);
	//DeviceMemOut("D:\\cuPhaseMap2.256.256.raw", cuPhaseMap2, Nx2, Ny2);
	//resize the Phase & Amp map
	bilinear_interpolation_kernel << <grid, block >> >(cuPhaseMap, cuPhaseMap2, Nx2, Ny2, Nx, Ny);
	//DeviceMemOut("D:\\cuPhaseMap.1024.1024.raw", cuPhaseMap, Nx, Ny);
	//find out the maximum and its index
	thrust::device_ptr<float> ptr = thrust::device_pointer_cast(cuPhaseMap);
	thrust::device_ptr<float> max_result = thrust::max_element(ptr, ptr + Nx*Ny);
	thrust::device_ptr<float> min_result = thrust::min_element(ptr, ptr + Nx*Ny);
	float maxPhi = max_result[0];
	float minPhi = min_result[0];

	//test
	/*float *tmpFloat = (float *)malloc(Nx*Ny*sizeof(float));
	hipMemcpy(tmpFloat, cuPhaseMap, sizeof(float)*Nx*Ny, hipMemcpyDeviceToHost);
	FILE *fp = fopen("D:\\buffer.raw", "wb");
	fwrite(tmpFloat, 1024 * 1024, sizeof(float), fp);
	fclose(fp);*/

	//convert to 1 byte	
	convert2oneByte << <grid, block >> >(tmp_uint8, cuPhaseMap, maxPhi, minPhi, maxPhi - minPhi, Nx, Ny);
	hipMemcpy(dst, tmp_uint8, sizeof(uint8_t)*Nx*Ny, hipMemcpyDeviceToHost);
}
//--------------------------------------------------------------------------------------
bool is_nan(double dVal)
{
	double dNan = std::numeric_limits<double>::quiet_NaN();

	if (dVal == dNan)
		return true;
	return false;
}
//--------------------------------------------------------------------------------------
bool is_inf(double dVal)
{
	double dNan = std::numeric_limits<double>::infinity();

	if (dVal == dNan)
		return true;
	return false;
}
//--------------------------------------------------------------------------------------
void DeviceMemOut(char *path, float *arr, int sizeX, int sizeY)
{
	int size = sizeX*sizeY;
	float *temp = (float *)malloc(size*sizeof(float));
	hipMemcpy(temp, arr, size*sizeof(float), hipMemcpyDeviceToHost);

	FILE *fp;
	fp = fopen(path, "wb");
	fwrite(temp, size, sizeof(float), fp);
	fclose(fp);
	free(temp);
}
//--------------------------------------------------------------------------------------
void DeviceMemOutFFT(char *path, hipfftComplex *arr, int sizeX, int sizeY)
{
	int size = sizeX*sizeY;
	hipfftComplex *temp = (hipfftComplex *)malloc(size*sizeof(hipfftComplex));
	float *temp2 = (float *)malloc(size*sizeof(float));
	hipMemcpy(temp, arr, size*sizeof(hipfftComplex), hipMemcpyDeviceToHost);

	for (int i = 0; i < size; i++)
	{
		temp2[i] = log10(sqrt(temp[i].x*temp[i].x + temp[i].y*temp[i].y));
		if (is_nan(temp2[i]) == true || is_inf(temp2[i]) == true) temp2[i] = 0;
	}

	FILE *fp;
	fp = fopen(path, "wb");
	fwrite(temp2, size, sizeof(float), fp);
	fclose(fp);
	free(temp);
	free(temp2);
}